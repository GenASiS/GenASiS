#include "hip/hip_runtime.h"
#include "DirectDevice.h"

#ifdef __cplusplus
extern "C"
  { 
#endif
void ComputeConservedPressureless_C
       ( double *D, double *S_1, double *S_2, double *S_3, 
         double *N, double *V_1, double *V_2, double *V_3, int nValues );
void ComputePrimitivePressureless_C
       ( double *N, double *V_1, double *V_2, double *V_3, 
         double *D, double *S_1, double *S_2, double *S_3, int nValues );
void ComputeEigenspeedsPressureless_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *V_1, double *V_2, double *V_3, int nValues );
void ApplyBoundaryConditionsReflectingPressureless_C
       ( double *N_E, double *VI_E, double *VJ_E, double *VK_E, 
	       double *N_I, double *VI_I, double *VJ_I, double *VK_I,
         int *nB, int *oBE, int *oBI, int *nSizes );
void ComputeRiemannSolverInputPressureless_C
       ( double *AP_I, double *AP_O, double *AM_I, double *AM_O,
         double *LP_I, double *LP_O, double *LM_I, double *LM_O,
         int *lV, int *uV, int *iaS_M, int *iaS_P, int *nSizes );
#ifdef __cplusplus
  }
#endif


__global__ void ComputeConservedPressurelessDeviceKernel 
                  ( double *D, double *S_1, double *S_2, double *S_3, 
                    double *N, double *V_1, double *V_2, double *V_3, int nValues )
  {
  if ( tiD < nValues )
    {	
    D   [ tiD ] = N [ tiD ];
    S_1 [ tiD ] = N [ tiD ] * V_1 [ tiD ];
    S_2 [ tiD ] = N [ tiD ] * V_2 [ tiD ];
    S_3 [ tiD ] = N [ tiD ] * V_3 [ tiD ]; 
    }
  }


void ComputeConservedPressureless_C
       ( double *D, double *S_1, double *S_2, double *S_3, 
         double *N, double *V_1, double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( ComputeConservedPressurelessDeviceKernel ), grid_Dim, block_Dim, 0, 0,
      D, S_1, S_2, S_3, N, V_1, V_2, V_3, nValues );
  DeviceSynchronize ( );
  }


__global__ void ComputePrimitivePressurelessDeviceKernel
	                ( double *N, double *V_1, double *V_2, double *V_3, 
	                  double *D, double *S_1, double *S_2, double *S_3, int nValues )
  { 
  if ( tiD < nValues )
    {
    N [ tiD ]  = D [ tiD ];
    if ( N [ tiD ] > 0.0 )
      {
      V_1 [ tiD ] = S_1 [ tiD ] / N [ tiD ];
      V_2 [ tiD ] = S_2 [ tiD ] / N [ tiD ];
      V_3 [ tiD ] = S_3 [ tiD ] / N [ tiD ];
      }
     else
      {
      N   [ tiD ] = 0.0;
      V_1 [ tiD ] = 0.0;
      V_2 [ tiD ] = 0.0;
      V_3 [ tiD ] = 0.0;
      D   [ tiD ] = 0.0;
      S_1 [ tiD ] = 0.0;
      S_2 [ tiD ] = 0.0;
      S_3 [ tiD ] = 0.0;
      }
    }
  }


void ComputePrimitivePressureless_C
       ( double *N, double *V_1, double *V_2, double *V_3, 
         double *D, double *S_1, double *S_2, double *S_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL
    ( ( ComputePrimitivePressurelessDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      N, V_1, V_2, V_3, D, S_1, S_2, S_3, nValues );
  DeviceSynchronize ( );
  }


__global__ void ComputeEigenspeedsPressurelessDeviceKernel
	                ( double *FEP_1, double *FEP_2, double *FEP_3,
	                  double *FEM_1, double *FEM_2, double *FEM_3,
	                  double *V_1, double *V_2, double *V_3, int nValues )
  {
  if ( tiD < nValues )
    {
    FEP_1 [ tiD ] = V_1 [ tiD ];
    FEP_2 [ tiD ] = V_2 [ tiD ];
    FEP_3 [ tiD ] = V_3 [ tiD ];
    FEM_1 [ tiD ] = V_1 [ tiD ];
    FEM_2 [ tiD ] = V_2 [ tiD ];
    FEM_3 [ tiD ] = V_3 [ tiD ];
    }
  }


void ComputeEigenspeedsPressureless_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *V_1, double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( ComputeEigenspeedsPressurelessDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      FEP_1, FEP_2, FEP_3, FEM_1, FEM_2, FEM_3, 
      V_1, V_2, V_3, nValues ); 
  DeviceSynchronize ( );
  }


__global__ void ApplyBoundaryConditionsReflectingPressureless
                  ( double *N_E, double *VI_E, double *VJ_E, double *VK_E, 
                    double *N_I, double *VI_I, double *VJ_I, double *VK_I,
                    int *nB, int *oBE, int *oBI, int iSize, int jSize, 
                    int kSize )
  {
  int iV, jV, kV, iI_I, jI_I, kI_I, iI_E, jI_E, kI_E;  // base-1 indexing
  int cI_I, cI_E; // 1D index for Interior and Exterior, base-0 indexing
 
  iV = ( tiD ) % nB [ 1 - 1 ] + 1;
  jV = ( tiD /  nB [ 1 - 1 ] ) % nB [ 2 - 1 ] + 1; 
  kV = tiD / ( nB [ 1 - 1 ] * nB [ 2 - 1 ] ) + 1;
  
  iI_I = oBI [ 0 ] + iV;
  jI_I = oBI [ 1 ] + jV;
  kI_I = oBI [ 2 ] + kV;
	
  cI_I = ( iI_I  +  iSize * ( jI_I - 1 )  
                 +  iSize * jSize * ( kI_I - 1 ) ) - 1;
	
  iI_E = oBE [ 0 ] + iV;
  jI_E = oBE [ 1 ] + jV;
  kI_E = oBE [ 2 ] + kV;
	
  cI_E = ( iI_E  +  iSize * ( jI_E - 1 )  
	               +  iSize * jSize * ( kI_E - 1 ) ) - 1;

  if ( kV <= nB [ 3 - 1 ] && jV <= nB [ 2 - 1 ] && iV <= nB [ 1 - 1 ] )
    {
    N_E  [ cI_E ] 	= N_I  [ cI_I ];
    VI_E [ cI_E ] 	= - VI_I [ cI_I ];
    VJ_E [ cI_E ] 	= VJ_I [ cI_I ];
    VK_E [ cI_E ] 	= VK_I [ cI_I ];
    }
  }


void ApplyBoundaryConditionsReflectingPressureless_C
       ( double *N_E, double *VI_E, double *VJ_E, double *VK_E, 
	       double *N_I, double *VI_I, double *VJ_I, double *VK_I,
         int *nB, int *oBE, int *oBI, int *nSizes )
  {
  int nValues = nSizes [ 3 - 1 ] * nSizes [ 2 - 1 ] * nSizes [ 1 - 1 ];
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );
  
  int *d_nB, *d_oBE, *d_oBI;
  
  hipHostMalloc ( &d_nB,  3 * sizeof ( int ) );
  hipHostMalloc ( &d_oBE, 3 * sizeof ( int ) );
  hipHostMalloc ( &d_oBI, 3 * sizeof ( int ) );
  
  hipMemcpy ( d_nB, nB, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_oBE, oBE, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_oBI, oBI, 3 * sizeof ( int ), hipMemcpyDefault );
  
  hipLaunchKernelGGL 
    ( ( ApplyBoundaryConditionsReflectingPressureless ),
      dim3 ( grid_Dim ), dim3 ( block_Dim ), 0, 0, 
	    N_E, VI_E, VJ_E, VK_E, N_I, VI_I, VJ_I, VK_I, d_nB, d_oBE, d_oBI, 
	    nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] ); 

  DeviceSynchronize (  ); 
  
  hipFree ( d_oBI );
  hipFree ( d_oBE );
  hipFree ( d_nB );
  }
  

__global__ void ComputeRiemannSolverInputPressurelessDeviceKernel
                  ( double *AP_I, double *AP_O, double *AM_I, double *AM_O,
                    double *LP_I, double *LP_O, double *LM_I, double *LM_O, 
                    int *lV, int *uV, int *iaS_M, int *iaS_P,
                    int iSize, int jSize, int kSize )
  {
  int iV, jV, kV, iaVS_M, jaVS_M, kaVS_M, iaVS_P, jaVS_P, kaVS_P;
  int A_iD, VS_MiD, VS_PiD; //1D indices
    
  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
   
  A_iD = ( iV + iSize * ( jV - 1 )
              + iSize * jSize * ( kV - 1 ) ) - 1;
  
  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS_M = iV + iaS_M [ 0 ];
        jaVS_M = jV + iaS_M [ 1 ];
        kaVS_M = kV + iaS_M [ 2 ];

        VS_MiD = ( iaVS_M + iSize * ( jaVS_M - 1 )
                          + iSize * jSize * ( kaVS_M - 1 ) ) - 1;
        
        iaVS_P = iV + iaS_P [ 0 ];
        jaVS_P = jV + iaS_P [ 1 ];
        kaVS_P = kV + iaS_P [ 2 ];
  
        VS_PiD = ( iaVS_P + iSize * ( jaVS_P - 1 )
                          + iSize * jSize * ( kaVS_P - 1 ) ) - 1; 
  
        AP_I [ A_iD ] 
          = max ( 0.0, max ( + LP_O [ VS_MiD ], + LP_I [ A_iD ] ) );
        AP_O [ A_iD ] 
          = max ( 0.0, max ( + LP_O [ A_iD ], + LP_I [ VS_PiD ] ) );
        AM_I [ A_iD ] 
          = max ( 0.0, max ( - LM_O [ VS_MiD ], - LM_I [ A_iD ] ) );
        AM_O [ A_iD ] 
          = max ( 0.0, max ( - LM_O [ A_iD ], - LM_I [ VS_PiD ] ) );
        }
      }
    }    
  }

      
void ComputeRiemannSolverInputPressureless_C
       ( double *AP_I, double *AP_O, double *AM_I, double *AM_O,
         double *LP_I, double *LP_O, double *LM_I, double *LM_O,
         int *lV, int *uV, int *iaS_M, int *iaS_P, int *nSizes )
  {
  int nValues = nSizes [ 3 - 1 ] * nSizes [ 2 - 1 ] * nSizes [ 1 - 1 ];
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ));
  
  int *d_lV, *d_uV, *d_iaS_M, *d_iaS_P; 
  
  hipHostMalloc ( &d_lV, 3 * sizeof ( int ) );
  hipHostMalloc ( &d_uV, 3 * sizeof ( int ) );
  hipHostMalloc ( &d_iaS_M, 3 * sizeof ( int ) );
  hipHostMalloc ( &d_iaS_P, 3 * sizeof ( int ) );
  
  hipMemcpy ( d_lV, lV, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_uV, uV, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_iaS_M, iaS_M, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_iaS_P, iaS_P, 3 * sizeof ( int ), hipMemcpyDefault );

  hipLaunchKernelGGL
    ( ( ComputeRiemannSolverInputPressurelessDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0,
      AP_I, AP_O, AM_I, AM_O, LP_I, LP_O, LM_I, LM_O, d_lV, d_uV, 
      d_iaS_M, d_iaS_P, nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );
  DeviceSynchronize ( );
  
  hipFree ( d_iaS_P );
  hipFree ( d_iaS_M );
  hipFree ( d_uV );
  hipFree ( d_lV );
  }
