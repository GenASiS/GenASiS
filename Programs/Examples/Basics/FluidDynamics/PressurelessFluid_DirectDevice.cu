#include "hip/hip_runtime.h"
#include "DirectDevice.h"

#ifdef __cplusplus
extern "C"
  { 
#endif
void ComputeConservedPressureless_C
       ( double *D, double *S_1, double *S_2, double *S_3, 
         double *N, double *V_1, double *V_2, double *V_3, int nValues );
void ComputePrimitivePressureless_C
       ( double *N, double *V_1, double *V_2, double *V_3, 
         double *D, double *S_1, double *S_2, double *S_3, int nValues );
void ComputeEigenspeedsPressureless_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *V_1, double *V_2, double *V_3, int nValues );
#ifdef __cplusplus
  }
#endif


__global__ void ComputeConservedPressurelessDeviceKernel 
                  ( double *D, double *S_1, double *S_2, double *S_3, 
                    double *N, double *V_1, double *V_2, double *V_3, int nValues )
  {
  if ( tiD < nValues )
    {	
    D   [ tiD ] = N [ tiD ];
    S_1 [ tiD ] = N [ tiD ] * V_1 [ tiD ];
    S_2 [ tiD ] = N [ tiD ] * V_2 [ tiD ];
    S_3 [ tiD ] = N [ tiD ] * V_3 [ tiD ]; 
    }
  }


void ComputeConservedPressureless_C
       ( double *D, double *S_1, double *S_2, double *S_3, 
         double *N, double *V_1, double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( ComputeConservedPressurelessDeviceKernel ), grid_Dim, block_Dim, 0, 0,
      D, S_1, S_2, S_3, N, V_1, V_2, V_3, nValues );
  DeviceSynchronize ( );
  }


__global__ void ComputePrimitivePressurelessDeviceKernel
	                ( double *N, double *V_1, double *V_2, double *V_3, 
	                  double *D, double *S_1, double *S_2, double *S_3, int nValues )
  { 
  if ( tiD < nValues )
    {
    N [ tiD ]  = D [ tiD ];
    if ( N [ tiD ] > 0.0 )
      {
      V_1 [ tiD ] = S_1 [ tiD ] / N [ tiD ];
      V_2 [ tiD ] = S_2 [ tiD ] / N [ tiD ];
      V_3 [ tiD ] = S_3 [ tiD ] / N [ tiD ];
      }
     else
      {
      N   [ tiD ] = 0.0;
      V_1 [ tiD ] = 0.0;
      V_2 [ tiD ] = 0.0;
      V_3 [ tiD ] = 0.0;
      D   [ tiD ] = 0.0;
      S_1 [ tiD ] = 0.0;
      S_2 [ tiD ] = 0.0;
      S_3 [ tiD ] = 0.0;
      }
    }
  }


void ComputePrimitivePressureless_C
       ( double *N, double *V_1, double *V_2, double *V_3, 
         double *D, double *S_1, double *S_2, double *S_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL
    ( ( ComputePrimitivePressurelessDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      N, V_1, V_2, V_3, D, S_1, S_2, S_3, nValues );
  DeviceSynchronize ( );
  }


__global__ void ComputeEigenspeedsPressurelessDeviceKernel
	                ( double *FEP_1, double *FEP_2, double *FEP_3,
	                  double *FEM_1, double *FEM_2, double *FEM_3,
	                  double *V_1, double *V_2, double *V_3, int nValues )
  {
  if ( tiD < nValues )
    {
    FEP_1 [ tiD ] = V_1 [ tiD ];
    FEP_2 [ tiD ] = V_2 [ tiD ];
    FEP_3 [ tiD ] = V_3 [ tiD ];
    FEM_1 [ tiD ] = V_1 [ tiD ];
    FEM_2 [ tiD ] = V_2 [ tiD ];
    FEM_3 [ tiD ] = V_3 [ tiD ];
    }
  }


void ComputeEigenspeedsPressureless_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *V_1, double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( ComputeEigenspeedsPressurelessDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      FEP_1, FEP_2, FEP_3, FEM_1, FEM_2, FEM_3, 
      V_1, V_2, V_3, nValues ); 
  DeviceSynchronize ( );
  }

