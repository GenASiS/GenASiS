#include "hip/hip_runtime.h"
#include "DirectDevice.h"

#ifdef __cplusplus
extern "C"
  { 
#endif
  void ComputeDifferences_C
         ( double *V, int *lV, int *uV, int *iaS, int iD, 
           double *dV_Left, double *dV_Right, int *nSizes );
  void ComputeReconstruction_C
       ( double *V, double *dV_Left, double *dV_Right, double Theta,
         double *V_Inner, double *V_Outer, int nValues );
  void ComputeFluxes_C
       ( double *AP_I, double *AP_O, double *AM_I, double *AM_O,
         double *RF_I, double *RF_O, double *U_I, double *U_O,  
         int *lV, int *uV, int *iaS, int iD, double *F_I, double *F_O,
         int *nSizes );
  void ComputeUpdate_C
         ( double *dU, double *F_I, double *F_O,
           double V, double A, double dT, int nValues );
  void AddUpdate_C ( double *O, double *U, double *C, int nValues );
  void CombineUpdates_C ( double *C, double *O, double *U, int nValues );
#ifdef __cplusplus
  }
#endif


__global__ void ComputeDifferencesLeftDeviceKernel
                  ( double *V, int *lV, int *uV, int *iaS, double *dV,
                    int iSize, int jSize, int kSize )
  { 
  int iV, jV, kV, iaVS, jaVS, kaVS;
  int dV_iD, VS_iD; //1D index
     
  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1 ;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
     
  dV_iD = ( iV + iSize * ( jV - 1 ) 
               + iSize * jSize * ( kV - 1 ) ) - 1;
  
  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS = iV + iaS [ 0 ];
        jaVS = jV + iaS [ 1 ];
        kaVS = kV + iaS [ 2 ];
  
        VS_iD = ( iaVS + iSize * ( jaVS - 1 )
                       + iSize * jSize * ( kaVS - 1 ) ) - 1;
   
        dV [ dV_iD ] = V [ dV_iD ] - V [ VS_iD ];
        }
      }
    }
  
  }

   
__global__ void ComputeDifferencesRightDeviceKernel
                  ( double *V, int *lV, int *uV, int *iaS, double *dV,
                    int iSize, int jSize, int kSize )
  {
  int iV, jV, kV, iaVS, jaVS, kaVS;
  int dV_iD, VS_iD; //1D index

  
  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
   
  dV_iD = ( iV + iSize * ( jV - 1 ) 
               + iSize * jSize * ( kV - 1 ) ) - 1;
  

  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS = iV + iaS [ 0 ];
        jaVS = jV + iaS [ 1 ];
        kaVS = kV + iaS [ 2 ];
    
        VS_iD = ( iaVS + iSize * ( jaVS - 1 )
                       + iSize * jSize * ( kaVS - 1 ) ) - 1;  
  
         dV [ dV_iD ] = V [ VS_iD ] - V [ dV_iD ];
        }
      }
    }
 
  }
 

void ComputeDifferences_C
       ( double *V, int *lV, int *uV, int *iaS, int iD, 
         double *dV_Left, double *dV_Right, int *nSizes )
  {
  int nValues = nSizes [ 3 - 1 ] * nSizes [ 2 - 1 ] * nSizes [ 1 - 1 ];
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
   
  iaS [ iD - 1 ] = -1;
  hipLaunchKernelGGL 
    ( ( ComputeDifferencesLeftDeviceKernel ), 
      dim3 ( grid_Dim ), dim3 ( block_Dim ), 0, 0, 
      V, lV, uV, iaS, dV_Left, nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );
  
  DeviceSynchronize (  );
  
  iaS [ iD - 1 ] = +1;
  hipLaunchKernelGGL 
    ( ( ComputeDifferencesRightDeviceKernel ), 
        dim3 ( grid_Dim ), dim3 ( block_Dim ), 0, 0,
      V, lV, uV, iaS, dV_Right, nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );

  DeviceSynchronize (  );
  }
  

__global__ void ComputeReconstructionDeviceKernel
                  ( double *V, double *dV_Left, double *dV_Right, 
                    double Theta, double *V_Inner, double *V_Outer, 
                    int nValues )
  {
  double dV;
  
  if ( tiD < nValues )
    {
    dV = ( copysign ( 0.5, dV_Left [ tiD ] )
             + copysign ( 0.5, dV_Right [ tiD ] ) )
         * min ( min ( abs ( Theta * dV_Left [ tiD ] ),
                       abs ( Theta * dV_Right [ tiD ] ) ),
                 abs ( 0.5 * ( dV_Left [ tiD ] + dV_Right [ tiD ] ) ) );
    V_Inner [ tiD ] = V [ tiD ] - 0.5 * dV;  
    V_Outer [ tiD ] = V [ tiD ] + 0.5 * dV;
    }
  }


void ComputeReconstruction_C
       ( double *V, double *dV_Left, double *dV_Right, double Theta,
         double *V_Inner, double *V_Outer, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
  
  hipLaunchKernelGGL
    ( ( ComputeReconstructionDeviceKernel ), grid_Dim, block_Dim, 0, 0,
        V, dV_Left, dV_Right, Theta, V_Inner, V_Outer, nValues );

  DeviceSynchronize ( );
  }
  
  
__global__ void ComputeFluxesInnerDeviceKernel
                  ( double *AP, double *AM, double *RF_I, double *RF_O, 
                    double *U_I, double *U_O, int *lV, int *uV, int *iaS,
                    double *F, int iSize, int jSize, int kSize )
  {
  int iV, jV, kV, iaVS, jaVS, kaVS;
  int F_iD, VS_iD; //1D index

  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1 ;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
   
  F_iD = ( iV + iSize * ( jV - 1 )
              + iSize * jSize * ( kV - 1 ) ) - 1;

  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS = iV + iaS [ 0 ];
        jaVS = jV + iaS [ 1 ];
        kaVS = kV + iaS [ 2 ];

        VS_iD = ( iaVS + iSize * ( jaVS - 1 )
                       + iSize * jSize * ( kaVS - 1 ) ) - 1;
  
        if ( AP [ F_iD ] + AM [ F_iD ] > 0.0 )
          {
          F [ F_iD ] = ( AP [ F_iD ] * RF_O [ VS_iD ]
                            + AM [ F_iD ] * RF_I [ F_iD ]
                            - AP [ F_iD ] * AM [ F_iD ]
                         * ( U_I [ F_iD ]  -  U_O [ VS_iD ] ) )
                       / ( AP [ F_iD ] + AM [ F_iD ] );
          }
        else
          {
          F [ F_iD ] = 0.0;
          }
        }
      }
    } 
  }


__global__ void ComputeFluxesOuterDeviceKernel
                  ( double *AP, double *AM, double *RF_I, double *RF_O,
                    double *U_I, double *U_O, int *lV, int *uV, int *iaS,
                    double *F, int iSize, int jSize, int kSize )
  {
  int iV, jV, kV, iaVS, jaVS, kaVS;
  int F_iD, VS_iD; //1D index
    
    
  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
   
  F_iD = ( iV + iSize * ( jV - 1 )
              + iSize * jSize * ( kV - 1 ) ) - 1;
  
  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS = iV + iaS [ 0 ];
        jaVS = jV + iaS [ 1 ];
        kaVS = kV + iaS [ 2 ];
  
        VS_iD = ( iaVS + iSize * ( jaVS - 1 )
                       + iSize * jSize * ( kaVS - 1 ) ) - 1;
  
        if ( AP [ F_iD ] + AM [ F_iD ] > 0.0 )
          {
          F  [ F_iD ] = ( AP [ F_iD ] * RF_O [ F_iD ]
                             + AM [ F_iD ] * RF_I [ VS_iD ]
                             - AP [ F_iD ] * AM [ F_iD ]   
                          * ( U_I [ VS_iD ]  -  U_O [ F_iD ] ) )
                        / ( AP [ F_iD ] + AM [ F_iD ] );
          }
        else
          {
          F [ F_iD ] = 0.0;
          }
        }
      }  
    }    
  
  }      


void ComputeFluxes_C
       ( double *AP_I, double *AP_O, double *AM_I, double *AM_O,
         double *RF_I, double *RF_O, double *U_I, double *U_O,  
         int *lV, int *uV, int *iaS, int iD, double *F_I, double *F_O,
         int *nSizes ) 
  
  {
  int nValues = nSizes [ 3 - 1 ] * nSizes [ 2 - 1 ] * nSizes [ 1 - 1 ];
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
  
  iaS [ iD - 1 ] = -1;
  hipLaunchKernelGGL 
    ( ( ComputeFluxesInnerDeviceKernel ), grid_Dim, block_Dim, 0, 0,
      AP_I, AM_I, RF_I, RF_O, U_I, U_O, lV, uV, iaS, F_I, 
      nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] ); 
  
  DeviceSynchronize (  );
  
  iaS [ iD - 1 ] = +1;
  hipLaunchKernelGGL
    ( ( ComputeFluxesOuterDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      AP_O, AM_O, RF_I, RF_O, U_I, U_O, lV, uV, iaS, F_O, 
      nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );
      
  DeviceSynchronize (  );
  }
                                                           
  
__global__ void ComputeUpdateDeviceKernel
                  ( double *dU, double *F_I, double *F_O,
                    double V, double A, double dT, int nValues )
  {
  if ( tiD <  nValues )
    dU [ tiD ] = dU [ tiD ] - dT * ( F_O [ tiD ] - F_I [ tiD ] ) * ( A / V );
  }


void ComputeUpdate_C
       ( double *dU, double *F_I, double *F_O,
         double V, double A, double dT, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
  
  hipLaunchKernelGGL
    ( ( ComputeUpdateDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      dU, F_I, F_O, V, A, dT, nValues );

  DeviceSynchronize ( );
  }


__global__ void AddUpdateDeviceKernel
                  ( double *O, double *U, double *C, int nValues )
  {
  if ( tiD < nValues )
    C [ tiD ] = O [ tiD ] + U [ tiD ];
  }
                

void AddUpdate_C ( double *O, double *U, double *C, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( AddUpdateDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      O, U, C, nValues );
    
  DeviceSynchronize ( );
  }


__global__ void CombineUpdatesDeviceKernel
                  ( double *C, double *O, double *U, int nValues )
  {
  if ( tiD < nValues )
    C [ tiD ] = 0.5 * ( O [ tiD ] + ( C [ tiD ] + U [ tiD ] ) );
  }
  
void CombineUpdates_C ( double *C, double *O, double *U, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
    
  hipLaunchKernelGGL 
    ( ( CombineUpdatesDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      C, O, U, nValues );
 
  DeviceSynchronize ( );
  }
