#include "hip/hip_runtime.h"
#include "DirectDevice.h"

#ifdef __cplusplus
extern "C"
  { 
#endif
  void ComputeDifferences_C
         ( double *V, int *lV, int *uV, int *iaS, int iD, 
           double *dV_Left, double *dV_Right, int *nSizes );
  void ComputeUpdate_C
         ( double *dU, double *F_I, double *F_O,
           double V, double A, double dT, int nValues );
  void AddUpdate_C ( double *O, double *U, double *C, int nValues );
  void CombineUpdates_C ( double *C, double *O, double *U, int nValues );
#ifdef __cplusplus
  }
#endif


__global__ void ComputeDifferencesLeftDeviceKernel
                  ( double *V, int *lV, int *uV, int *iaS, double *dV,
                    int iSize, int jSize, int kSize )
  { 
  int iV, jV, kV, iaVS, jaVS, kaVS;
  int dV_iD, VS_iD; //1D index
     
  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1 ;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
     
  dV_iD = ( iV + iSize * ( jV - 1 ) 
               + iSize * jSize * ( kV - 1 ) ) - 1;
  
  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS = iV + iaS [ 0 ];
        jaVS = jV + iaS [ 1 ];
        kaVS = kV + iaS [ 2 ];
  
        VS_iD = ( iaVS + iSize * ( jaVS - 1 )
                       + iSize * jSize * ( kaVS - 1 ) ) - 1;
   
        dV [ dV_iD ] = V [ dV_iD ] - V [ VS_iD ];
       }
     }
   }
  
 }

   
__global__ void ComputeDifferencesRightDeviceKernel
                  ( double *V, int *lV, int *uV, int *iaS, double *dV,
                    int iSize, int jSize, int kSize )
  {
  int iV, jV, kV, iaVS, jaVS, kaVS;
  int dV_iD, VS_iD; //1D index

  
  iV = ( tiD ) % ( uV [ 0 ] ) + 1;
  jV = ( tiD / uV [ 0 ] ) %  uV [ 1 ] + 1;
  kV = tiD / ( uV [ 0 ] *  uV [ 1 ] ) + 1;
   
  dV_iD = ( iV + iSize * ( jV - 1 ) 
               + iSize * jSize * ( kV - 1 ) ) - 1;
  

  if ( kV >= lV [ 2 ] && kV <= uV [ 2 ] )
    {
    if ( jV >= lV [ 1 ] && jV <= uV [ 1 ] )
      {
      if ( iV >= lV [ 0 ] && iV <= uV [ 0 ] )
        { 
        iaVS = iV + iaS [ 0 ];
        jaVS = jV + iaS [ 1 ];
        kaVS = kV + iaS [ 2 ];
    
        VS_iD = ( iaVS + iSize * ( jaVS - 1 )
                       + iSize * jSize * ( kaVS - 1 ) ) - 1;  
  
         dV [ dV_iD ] = V [ VS_iD ] - V [ dV_iD ];
        }
      }
    }
 
  }
 

void ComputeDifferences_C
       ( double *V, int *lV, int *uV, int *iaS, int iD, 
         double *dV_Left, double *dV_Right, int *nSizes )
  {
  int nValues = nSizes [ 3 - 1 ] * nSizes [ 2 - 1 ] * nSizes [ 1 - 1 ];
  /*
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );
  */
  int block_Dim = 256;
  int grid_Dim  = ceil ( nValues * 1.0 ) / block_Dim;
   
  iaS [ iD - 1 ] = -1;
  
  printf ( "ComputeDifferences_C Left: %d %d\n", grid_Dim, block_Dim );
  printf ( "V : %p, dV_Left: %p\n", V, dV_Left );
  
  /*
  hipLaunchKernelGGL 
    ( ( ComputeDifferencesLeftDeviceKernel ), 
      dim3 ( grid_Dim ), dim3 ( block_Dim ), 0, 0, 
      V, lV, uV, iaS, dV_Left, nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );
  
  DeviceSynchronize (  );
  
  hipError_t hipErrSync  = hipGetLastError();
  if (hipErrSync != hipSuccess)
    { 
    printf("HIP Error - %s:%d: '%s'\n", __FILE__, __LINE__,
    hipGetErrorString(hipErrSync)); exit(0);
    }
  */    
  iaS [ iD - 1 ] = +1;
  
  printf ( "ComputeDifferences_C Right\n" );
  /*
  hipLaunchKernelGGL 
    ( ( ComputeDifferencesRightDeviceKernel ), 
        dim3 ( grid_Dim ), dim3 ( block_Dim ), 0, 0,
      V, lV, uV, iaS, dV_Right, nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );
  */
  //DeviceSynchronize (  );
  }
  
__global__ void ComputeUpdateDeviceKernel
                  ( double *dU, double *F_I, double *F_O,
                    double V, double A, double dT, int nValues )
  {
  if ( tiD <  nValues )
    dU [ tiD ] = dU [ tiD ] - dT * ( F_O [ tiD ] - F_I [ tiD ] ) * ( A / V );
  }


void ComputeUpdate_C
       ( double *dU, double *F_I, double *F_O,
         double V, double A, double dT, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
  
  hipLaunchKernelGGL
    ( ( ComputeUpdateDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      dU, F_I, F_O, V, A, dT, nValues );

  DeviceSynchronize ( );
  }


__global__ void AddUpdateDeviceKernel
                  ( double *O, double *U, double *C, int nValues )
  {
  if ( tiD < nValues )
    C [ tiD ] = O [ tiD ] + U [ tiD ];
  }
                

void AddUpdate_C ( double *O, double *U, double *C, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( AddUpdateDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      O, U, C, nValues );
    
  DeviceSynchronize ( );
  }


__global__ void CombineUpdatesDeviceKernel
                  ( double *C, double *O, double *U, int nValues )
  {
  if ( tiD < nValues )
    C [ tiD ] = 0.5 * ( O [ tiD ] + ( C [ tiD ] + U [ tiD ] ) );
  }
  
void CombineUpdates_C ( double *C, double *O, double *U, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x - 1 ) / block_Dim.x ) );
    
  hipLaunchKernelGGL 
    ( ( CombineUpdatesDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      C, O, U, nValues );
 
  DeviceSynchronize ( );
  }
