#include "hip/hip_runtime.h"
#include "DirectDevice.h"

#ifdef __cplusplus
extern "C"
  { 
#endif
void ComputeConservedPolytropic_C
       ( double *G, double *E, double *N, double *V_1, 
         double *V_2, double *V_3, int nValues );
void ComputePrimitivePolytropic_C
	     ( double *E, double *G, double *N, 
         double *V_1, double *V_2, double *V_3, int nValues );
void ComputeAuxiliaryPolytropic_C
       ( double *P, double *K, double *N, double *E, 
         double *Gamma, int nValues );
void ComputeEigenspeedsPolytropic_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *CS, double *N, double *V_1, double *V_2, 
         double *V_3, double *P, double *Gamma, int nValues );
void ApplyBoundaryConditionsReflectingPolytropic_C
       ( double *E_E, double *Gamma_E, double *E_I, double *Gamma_I,
         int *nB, int *oBE, int *oBI, int *nSizes );
void ComputeRawFluxesPolytropic_C
       ( double *F_D, double *F_S_1, double *F_S_2, double *F_S_3,
         double *F_S_Dim, double *F_G, double *D, double *S_1, double *S_2, 
         double *S_3, double *G, double *P, double *V_Dim, int nValues );
#ifdef __cplusplus
  }
#endif


__global__ void ComputeConservedPolytropicDeviceKernel 
                  ( double *G, double *E, double *N, double *V_1, 
                    double *V_2, double *V_3, int nValues )
  {
  if ( tiD < nValues )
    {
    G [ tiD ] = E [ tiD ] + 0.5 * N [ tiD ] 
                  * ( V_1 [ tiD ] * V_1 [ tiD ]
                        + V_2 [ tiD ] * V_2 [ tiD ] 
                        + V_3 [ tiD ] * V_3 [ tiD ] );
    }
  }


void ComputeConservedPolytropic_C
       ( double *G, double *E, double *N, double *V_1, 
         double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL
    ( ( ComputeConservedPolytropicDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      G, E, N, V_1, V_2, V_3, nValues );
  DeviceSynchronize ( );
  }
  

__global__ void ComputePrimitivePolytropicDeviceKernel
                  ( double *E, double *G, double *N, double *V_1, 
                    double *V_2, double *V_3, int nValues )
  {
  double KE;  	
	
  if ( tiD < nValues )
    {
    KE = 0.5 * N [ tiD ] 
           * ( V_1 [ tiD ] * V_1 [ tiD ]
                 + V_2 [ tiD ] * V_2 [ tiD ]
                 + V_3 [ tiD ] * V_3 [ tiD ] );

    E [ tiD ] = G [ tiD ] - KE;
    if ( E [ tiD ] < 0.0 )
      {
      E [ tiD ] = 0.0;
      G [ tiD ] = KE; 
      }
    }
  }


void ComputePrimitivePolytropic_C
	     ( double *E, double *G, double *N, 
         double *V_1, double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL
    ( ( ComputePrimitivePolytropicDeviceKernel ), grid_Dim, block_Dim, 0, 0,
      E, G, N, V_1, V_2, V_3, nValues );
  DeviceSynchronize ( );
  }


__global__ void ComputeAuxiliaryPolytropicDeviceKernel 
                  ( double *P, double *K, double *N, double *E, 
                    double *Gamma, int nValues )
  {
  if ( tiD < nValues )
    {
    P [ tiD ] = E [ tiD ] * ( Gamma [ tiD ] - 1.0 );
    if ( N [ tiD ] > 0.0 )
      K [ tiD ] = P [ tiD ] / pow ( N [ tiD ], Gamma [ tiD ] );
    else
      K [ tiD ] = 0.0;
    }
  }


void ComputeAuxiliaryPolytropic_C
       ( double *P, double *K, double *N, double *E, 
         double *Gamma, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ));

  hipLaunchKernelGGL 
    ( ( ComputeAuxiliaryPolytropicDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0,
      P, K, N, E, Gamma, nValues ); 
  DeviceSynchronize ( );
  }
  

__global__ void ComputeEigenspeedsPolytropicDeviceKernel
                  ( double *FEP_1, double *FEP_2, double *FEP_3,
                    double *FEM_1, double *FEM_2, double *FEM_3,
                    double *CS, double *N, double *V_1, double *V_2, 
                    double *V_3, double *P,  double *Gamma, int nValues )
  {
  if ( tiD < nValues )
    {
    if ( N [ tiD ] > 0.0 && P [ tiD ] > 0.0 )
      CS [ tiD ] = sqrt ( Gamma [ tiD ] * P [ tiD ] / N [ tiD ] );
    else
      CS [ tiD ] = 0.0;

    FEP_1 [ tiD ] = V_1 [ tiD ] + CS [ tiD ];
    FEP_2 [ tiD ] = V_2 [ tiD ] + CS [ tiD ];
    FEP_3 [ tiD ] = V_3 [ tiD ] + CS [ tiD ];
    FEM_1 [ tiD ] = V_1 [ tiD ] - CS [ tiD ];
    FEM_2 [ tiD ] = V_2 [ tiD ] - CS [ tiD ];
    FEM_3 [ tiD ] = V_3 [ tiD ] - CS [ tiD ];
    }
  
  }
      

void ComputeEigenspeedsPolytropic_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *CS, double *N, double *V_1, double *V_2, 
         double *V_3, double *P, double *Gamma, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( ComputeEigenspeedsPolytropicDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      FEP_1, FEP_2, FEP_3, FEM_1, FEM_2, FEM_3,
      CS, N, V_1, V_2, V_3, P, Gamma, nValues ); 
  DeviceSynchronize ( );
  }


__global__ void ApplyBoundaryConditionsReflectingPolytropic
                  ( double *E_E, double *Gamma_E,
                    double *E_I, double *Gamma_I,
                    int *nB, int *oBE, int *oBI, 
                    int iSize, int jSize, int kSize )
  {
  int iV, jV, kV, iI_I, jI_I, kI_I, iI_E, jI_E, kI_E;  // base-1 indexing
  int cI_I, cI_E; // 1D index for Interior and Exterior, base-0 indexing 
 
  iV = ( tiD ) % nB [ 1 - 1 ] + 1;
  jV = ( tiD /  nB [ 1 - 1 ] ) % nB [ 2 - 1 ] + 1;
  kV = tiD / ( nB [ 1 - 1 ] * nB [ 2 - 1 ] ) + 1; 
 
  iI_I = oBI [ 0 ] + iV;
  jI_I = oBI [ 1 ] + jV;
  kI_I = oBI [ 2 ] + kV;
  
  cI_I = ( iI_I  +  iSize * ( jI_I - 1 )
                 +  iSize * jSize * ( kI_I - 1 ) ) - 1;
  
  iI_E = oBE [ 0 ] + iV;
  jI_E = oBE [ 1 ] + jV;
  kI_E = oBE [ 2 ] + kV;
  
  cI_E = ( iI_E  +  iSize * ( jI_E - 1 )
                 +  iSize * jSize * ( kI_E - 1 ) ) - 1;
  
  if ( kV <= nB [ 3 - 1 ]  &&  jV <= nB [ 2 - 1 ]  &&  iV <= nB [ 1 - 1 ] )
    {
    E_E     [ cI_E ]  = E_I     [ cI_I ];
    Gamma_E [ cI_E ]  = Gamma_I [ cI_I ];
    }
  }

     
void ApplyBoundaryConditionsReflectingPolytropic_C
       ( double *E_E, double *Gamma_E, double *E_I, double *Gamma_I,
         int *nB, int *oBE, int *oBI, int *nSizes )
  {
  int nValues = nSizes [ 3 - 1 ] * nSizes [ 2 - 1 ] * nSizes [ 1 - 1 ];
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );
  
  int *d_nB, *d_oBE, *d_oBI;
  
  /*
  hipHostMalloc ( &d_nB,  3 * sizeof ( int ) );
  hipHostMalloc ( &d_oBE, 3 * sizeof ( int ) );
  hipHostMalloc ( &d_oBI, 3 * sizeof ( int ) );
  */
  
  hipMalloc ( &d_nB,  3 * sizeof ( int ) );
  hipMalloc ( &d_oBE, 3 * sizeof ( int ) );
  hipMalloc ( &d_oBI, 3 * sizeof ( int ) );
  
  hipMemcpy ( d_nB, nB, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_oBE, oBE, 3 * sizeof ( int ), hipMemcpyDefault );
  hipMemcpy ( d_oBI, oBI, 3 * sizeof ( int ), hipMemcpyDefault );
  
  hipLaunchKernelGGL
    ( ( ApplyBoundaryConditionsReflectingPolytropic ), 
      grid_Dim, block_Dim, 0, 0,
      E_E, Gamma_E, E_I, Gamma_I, d_nB, d_oBE, d_oBI, 
      nSizes [ 0 ], nSizes [ 1 ], nSizes [ 2 ] );
  
  DeviceSynchronize (  );
  
  hipFree ( d_oBI );
  hipFree ( d_oBE );
  hipFree ( d_nB );
  }
 

__global__ void ComputeRawFluxesPolytropicDeviceKernel
                  ( double *F_D, double *F_S_1, double *F_S_2, 
                    double *F_S_3, double *F_S_Dim, double *F_G,
                    double *D, double *S_1, double *S_2, double *S_3,
                    double *G, double *P, double *V_Dim, int nValues )
  {
  if ( tiD < nValues )
    {
    F_D [ tiD ]     = D [ tiD ]   * V_Dim [ tiD ];
    F_S_1 [ tiD ]   = S_1 [ tiD ] * V_Dim [ tiD ];
    F_S_2 [ tiD ]   = S_2 [ tiD ] * V_Dim [ tiD ];
    F_S_3 [ tiD ]   = S_3 [ tiD ] * V_Dim [ tiD ];
    F_S_Dim [ tiD ] = F_S_Dim [ tiD ] + P [ tiD ];
    F_G [ tiD ]     = ( G [ tiD ] + P [ tiD ] ) * V_Dim [ tiD ];
    } 
  }


void ComputeRawFluxesPolytropic_C
       ( double *F_D, double *F_S_1, double *F_S_2, double *F_S_3,
         double *F_S_Dim, double *F_G, double *D, double *S_1, double *S_2, 
         double *S_3, double *G, double *P, double *V_Dim, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ));

  hipLaunchKernelGGL
    ( ( ComputeRawFluxesPolytropicDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      F_D, F_S_1, F_S_2, F_S_3, F_S_Dim, F_G, D, S_1, S_2, S_3, G, P, 
      V_Dim, nValues ); 
  DeviceSynchronize ( );
  }
