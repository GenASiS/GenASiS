#include "hip/hip_runtime.h"
#include "DirectDevice.h"

#ifdef __cplusplus
extern "C"
  { 
#endif
void ComputeConservedPolytropic_C
       ( double *G, double *E, double *N, double *V_1, 
         double *V_2, double *V_3, int nValues );
void ComputePrimitivePolytropic_C
	     ( double *E, double *G, double *N, 
         double *V_1, double *V_2, double *V_3, int nValues );
void ComputeAuxiliaryPolytropic_C
       ( double *P, double *K, double *N, double *E, 
         double *Gamma, int nValues );
void ComputeEigenspeedsPolytropic_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *CS, double *N, double *V_1, double *V_2, 
         double *V_3, double *P, double *Gamma, int nValues );
void ComputeRawFluxesPolytropic_C
       ( double *F_D, double *F_S_1, double *F_S_2, double *F_S_3,
         double *F_S_Dim, double *F_G, double *D, double *S_1, double *S_2, 
         double *S_3, double *G, double *P, double *V_Dim, int nValues );
#ifdef __cplusplus
  }
#endif


__global__ void ComputeConservedPolytropicDeviceKernel 
                  ( double *G, double *E, double *N, double *V_1, 
                    double *V_2, double *V_3, int nValues )
  {
  if ( tiD < nValues )
    {
    G [ tiD ] = E [ tiD ] + 0.5 * N [ tiD ] 
                  * ( V_1 [ tiD ] * V_1 [ tiD ]
                        + V_2 [ tiD ] * V_2 [ tiD ] 
                        + V_3 [ tiD ] * V_3 [ tiD ] );
    }
  }


void ComputeConservedPolytropic_C
       ( double *G, double *E, double *N, double *V_1, 
         double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL
    ( ( ComputeConservedPolytropicDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      G, E, N, V_1, V_2, V_3, nValues );
  DeviceSynchronize ( );
  }
  

__global__ void ComputePrimitivePolytropicDeviceKernel
                  ( double *E, double *G, double *N, double *V_1, 
                    double *V_2, double *V_3, int nValues )
  {
  double KE;  	
	
  if ( tiD < nValues )
    {
    KE = 0.5 * N [ tiD ] 
           * ( V_1 [ tiD ] * V_1 [ tiD ]
                 + V_2 [ tiD ] * V_2 [ tiD ]
                 + V_3 [ tiD ] * V_3 [ tiD ] );

    E [ tiD ] = G [ tiD ] - KE;
    if ( E [ tiD ] < 0.0 )
      {
      E [ tiD ] = 0.0;
      G [ tiD ] = KE; 
      }
    }
  }


void ComputePrimitivePolytropic_C
	     ( double *E, double *G, double *N, 
         double *V_1, double *V_2, double *V_3, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL
    ( ( ComputePrimitivePolytropicDeviceKernel ), grid_Dim, block_Dim, 0, 0,
      E, G, N, V_1, V_2, V_3, nValues );
  DeviceSynchronize ( );
  }


__global__ void ComputeAuxiliaryPolytropicDeviceKernel 
                  ( double *P, double *K, double *N, double *E, 
                    double *Gamma, int nValues )
  {
  if ( tiD < nValues )
    {
    P [ tiD ] = E [ tiD ] * ( Gamma [ tiD ] - 1.0 );
    if ( N [ tiD ] > 0.0 )
      K [ tiD ] = P [ tiD ] / pow ( N [ tiD ], Gamma [ tiD ] );
    else
      K [ tiD ] = 0.0;
    }
  }


void ComputeAuxiliaryPolytropic_C
       ( double *P, double *K, double *N, double *E, 
         double *Gamma, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ));

  hipLaunchKernelGGL 
    ( ( ComputeAuxiliaryPolytropicDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0,
      P, K, N, E, Gamma, nValues ); 
  DeviceSynchronize ( );
  }
  

__global__ void ComputeEigenspeedsPolytropicDeviceKernel
                  ( double *FEP_1, double *FEP_2, double *FEP_3,
                    double *FEM_1, double *FEM_2, double *FEM_3,
                    double *CS, double *N, double *V_1, double *V_2, 
                    double *V_3, double *P,  double *Gamma, int nValues )
  {
  if ( tiD < nValues )
    {
    if ( N [ tiD ] > 0.0 && P [ tiD ] > 0.0 )
      CS [ tiD ] = sqrt ( Gamma [ tiD ] * P [ tiD ] / N [ tiD ] );
    else
      CS [ tiD ] = 0.0;

    FEP_1 [ tiD ] = V_1 [ tiD ] + CS [ tiD ];
    FEP_2 [ tiD ] = V_2 [ tiD ] + CS [ tiD ];
    FEP_3 [ tiD ] = V_3 [ tiD ] + CS [ tiD ];
    FEM_1 [ tiD ] = V_1 [ tiD ] - CS [ tiD ];
    FEM_2 [ tiD ] = V_2 [ tiD ] - CS [ tiD ];
    FEM_3 [ tiD ] = V_3 [ tiD ] - CS [ tiD ];
    }
  
  }
      

void ComputeEigenspeedsPolytropic_C
       ( double *FEP_1, double *FEP_2, double *FEP_3,
         double *FEM_1, double *FEM_2, double *FEM_3,
         double *CS, double *N, double *V_1, double *V_2, 
         double *V_3, double *P, double *Gamma, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ) );

  hipLaunchKernelGGL 
    ( ( ComputeEigenspeedsPolytropicDeviceKernel ), 
      grid_Dim, block_Dim, 0, 0, 
      FEP_1, FEP_2, FEP_3, FEM_1, FEM_2, FEM_3,
      CS, N, V_1, V_2, V_3, P, Gamma, nValues ); 
  DeviceSynchronize ( );
  }


__global__ void ComputeRawFluxesPolytropicDeviceKernel
                  ( double *F_D, double *F_S_1, double *F_S_2, 
                    double *F_S_3, double *F_S_Dim, double *F_G,
                    double *D, double *S_1, double *S_2, double *S_3,
                    double *G, double *P, double *V_Dim, int nValues )
  {
  if ( tiD < nValues )
    {
    F_D [ tiD ]     = D [ tiD ]   * V_Dim [ tiD ];
    F_S_1 [ tiD ]   = S_1 [ tiD ] * V_Dim [ tiD ];
    F_S_2 [ tiD ]   = S_2 [ tiD ] * V_Dim [ tiD ];
    F_S_3 [ tiD ]   = S_3 [ tiD ] * V_Dim [ tiD ];
    F_S_Dim [ tiD ] = F_S_Dim [ tiD ] + P [ tiD ];
    F_G [ tiD ]     = ( G [ tiD ] + P [ tiD ] ) * V_Dim [ tiD ];
    } 
  }


void ComputeRawFluxesPolytropic_C
       ( double *F_D, double *F_S_1, double *F_S_2, double *F_S_3,
         double *F_S_Dim, double *F_G, double *D, double *S_1, double *S_2, 
         double *S_3, double *G, double *P, double *V_Dim, int nValues )
  {
  dim3 block_Dim ( BLOCK_DIM * BLOCK_DIM );
  dim3 grid_Dim  ( ceil ( ( nValues + block_Dim.x -1 ) / block_Dim.x ));

  hipLaunchKernelGGL
    ( ( ComputeRawFluxesPolytropicDeviceKernel ), grid_Dim, block_Dim, 0, 0, 
      F_D, F_S_1, F_S_2, F_S_3, F_S_Dim, F_G, D, S_1, S_2, S_3, G, P, 
      V_Dim, nValues ); 
  DeviceSynchronize ( );
  }
